#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>



unsigned int getmax(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
hipDeviceProp_t prop;
hipError_t error = hipGetDeviceProperties(&prop, 0);
	printf("char name[256]; = %s\n size_t totalGlobalMem; = %d\n size_t sharedMemPerBlock; = %d\n int regsPerBlock; = %d\n int warpSize; = %d\n size_t memPitch; = %d\n int maxThreadsPerBlock; = %d\n int maxThreadsDim[3]; = %d\n int maxGridSize[3]; = %d\n int clockRate; = %d\n size_t totalConstMem; = %d\n   int major; = %d\n int minor; = %d\n size_t textureAlignment; = %d\n size_t texturePitchAlignment; = %d\n int deviceOverlap; = %d\n int multiProcessorCount; = %d\n int kernelExecTimeoutEnabled; = %d\n int integrated; = %d\n int canMapHostMemory; = %d\n int computeMode; = %d\n", prop->name,prop->totalGlobalMem, prop->sharedMemPerBlock, prop->regsPerBlock, prop->warpSize,            prop->memPitch, prop->maxThreadsPerBlock,  prop->maxThreadsDim[3], prop->maxGridSize[3],             prop->clockRate, prop->totalConstMem,prop->major,   prop->minor, prop->textureAlignment,   prop->texturePitchAlignment, prop->deviceOverlap, prop->multiProcessorCount,        prop->kernelExecTimeoutEnabled, prop->integrated, prop->canMapHostMemory, prop-> computeMode,            );
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
	unsigned int block;
	unsigned int thread;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
	   
   getCuda
   // printf(" The maximum number in the array is: %u\n", 
     //      getmax<<<block, thread>>>(cudaNumbers, cudaSize, cudaN));

    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
_global_ unsigned int getmax(unsigned int num[], unsigned int size, int n)
{
	if(id<size%n)
		starting = (threadId.x+blockId.x*blockDim.x)(size/n+1);
	else
		starting = (size%n)(size/n+1)+((threadId.x+blockId.x*blockDim.x)-size%n)(size/n);
	blockId.x*blockDim.x+threadId.x;
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}