#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

/*
This code was developed and tested on cuda1

Device 0:
name: GeForce GTX TITAN Black
Compute capability 3.5
total global memory(KB): 6228288
shared mem per block: 49152
regs per block: 65536
warp size: 32
max threads per block: 1024
max thread dim z:1024 y:1024 x:64
max grid size z:2147483647 y:65535 x:65535
clock rate(KHz):
total constant memory (bytes): 65536
multiprocessor count 15
integrated: 0
async engine count: 1
memory bus width: 384
memory clock rate (KHz): 3500000
L2 cache size (bytes): 1572864
max threads per SM: 2048

*/


__global__ void getmaxcu(unsigned int num[], unsigned int size, int n){
	/*if(id<size%n)
		starting = (threadIdx.x+blockId.x*blockDim.x)(size/n+1);
	else
		starting = (size%n)(size/n+1)+((threadId.x+blockId.x*blockDim.x)-size%n)(size/n);*/
		
	
	unsigned int tid = threadIdx.x;
	unsigned int gloid = blockIdx.x*blockDim.x+threadIdx.x;	
	unsigned int tSize = size/n;
	
	const unsigned int dim = blockDim.x;
	__device__ __shared__ int sdata[]; // shared data
	
	if(tid<size%n)
		tSize++;
	__syncthreads();	
	
	//each thread iterates over its section of the large array
	sdata[tid]=num[gloid];
	for(unsigned int i = 0; i < tSize; i++)
		if(num[gloid] > sdata[tid])
			sdata[tid]=num[gloid];
			
	__syncthreads();
	
	//get a block max by performing a tree-structured 
	//reduction akin to that depicted in slide 17 of 
	//the lecture 8 pp
	
	for(unsigned int stride = 1; stride<blockDim.x; stride*=2){
		if(tid%(2*stride)==0){
			if(sdata[tid]<sdata[tid+stride])
				sdata[tid]=sdata[tid+stride];
		}
		__syncthreads();
	}
	
	if(tid==0){//store the block maxes in global memory
		num[blockIdx.x]=sdata[0];
	}

	//return(num[0]);

}

int main(int argc, char *argv[])
{
	hipDeviceProp_t prop;
	hipError_t propErr = hipGetDeviceProperties(&prop, 0);
	
	
	if (propErr != hipSuccess) {
		printf("unable to get device properties\n");
	}
	
	
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
	unsigned int* cudaNumbers;
	unsigned int block;
	unsigned int thread;
	
	block = 30;
	thread = 1024;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

	

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
	   
	for(int i = 0; i < size; i++) {
		printf("%d ", numbers[i]);
    }
    printf("\n"); 
	 
	hipMalloc((void**)&cudaNumbers, (size * sizeof(unsigned int)));
	hipMemcpy(cudaNumbers, numbers, (size * sizeof(unsigned int)), hipMemcpyHostToDevice);
	
	unsigned int cudaSize=size;
	//hipMalloc((void**)&cudaSize, sizeof(unsigned int));
	//hipMemcpy(cudaSize, size, (sizeof(unsigned int)), hipMemcpyHostToDevice);
	
	
	unsigned int cudaN = block*thread;
	//unsigned int cudaN;
	//hipMalloc((void**)&cudaN, sizeof(unsigned int));
	//hipMemcpy(cudaN, n, (sizeof(unsigned int)), hipMemcpyHostToDevice);
		
	
	getmaxcu<<<block, thread>>>(cudaNumbers, cudaSize, cudaN);
	//cudaSize/thread;
   
    
	getmaxcu<<<1, block>>>(cudaNumbers, block, block);
	printf("%s\n", hipMemcpy(numbers, cudaNumbers, sizeof(unsigned int), hipMemcpyDeviceToHost));

	printf(" The maximum number in the array is: %u\n", numbers[0]);

    free(numbers);
	hipFree(cudaNumbers);
	//hipFree(cudaSize);
	//hipFree(cudaN);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/