#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define maxThreads 512
/*
This code was developed and tested on cuda3

*/


__global__ void getmaxcu(unsigned int num[], unsigned int size){		
	
	unsigned int tid = threadIdx.x;
	unsigned int gloid = blockIdx.x*blockDim.x+threadIdx.x;	
	
	__shared__ int sdata[maxThreads]; // shared data
	
	sdata[tid]=num[gloid];
	if(gloid>=size){
		sdata[threadIdx.x]=0;
	}
	
	/*if(n<size){
		int tSize = size/n;
		if(tid<(size%n)
			tSize++;
		for(int i; i<tSize; i++)
			if(sdata[tid]<num[glo
	}
	*/
	__syncthreads();
	
	//get a block max by performing a tree-structured 
	//reduction akin to that depicted in slide 18 of 
	//the lecture 8 pp
	
	for (int stride = blockDim.x / 2; stride > 0; stride = stride / 2) {
        if (tid < stride) {
            if (sdata[tid] < sdata[tid + stride]) {
                sdata[tid] = sdata[tid + stride];
            }
        }
        __syncthreads();
	}
	
	if(tid==0){//store the block maxes in global memory
		num[blockIdx.x]=sdata[0];
	}
}

int main(int argc, char *argv[])
{
	hipDeviceProp_t prop;
	hipError_t propErr = hipGetDeviceProperties(&prop, 0);
	
	
	if (propErr != hipSuccess) {
		printf("unable to get device properties\n");
	}
	
	
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
	unsigned int* cudaNumbers;
	unsigned int thread;
	unsigned int block;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);
    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

	if (size%maxThreads != 0) {
        size = (size/maxThreads+1)*maxThreads;
    } 

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
	   
	//for(int i = 0; i < size; i++) {
	//	printf("%d ", numbers[i]);
    //}
    //printf("\n"); 
	
	hipMalloc((void**)&cudaNumbers, (size * sizeof(unsigned int)));
	hipMemcpy(cudaNumbers, numbers, (size * sizeof(unsigned int)), hipMemcpyHostToDevice);
	
	unsigned int cudaSize=size;
	thread = maxThreads;
	block = size/thread;
	
	//getmaxcu<<<block, thread>>>(cudaNumbers, cudaSize);
	while(block>1){
		getmaxcu<<<block, thread>>>(cudaNumbers, cudaSize);
		cudaSize=cudaSize/thread;
		thread = block;
		block = cudaSize/thread;
	}
	getmaxcu<<<1, block>>>(cudaNumbers, block);
	
	hipMemcpy(numbers, cudaNumbers, sizeof(unsigned int), hipMemcpyDeviceToHost);//only copies back the max, which should be in the first element of the array
	printf(" The maximum number in the array is: %u\n", numbers[0]);

    free(numbers);
	hipFree(cudaNumbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/