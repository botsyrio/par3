#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define thread 1024
/*
This code was developed and tested on cuda3

*/


__global__ void getmaxcu(unsigned int num[], unsigned int size, int n){		
	
	unsigned int tid = threadIdx.x;
	unsigned int gloid = blockIdx.x*blockDim.x+threadIdx.x;	
	
	__shared__ int sdata[thread]; // shared data
	
	sdata[tid]=num[gloid];
	if(gloid>=size){
		sdata[threadIdx.x]=0;
	}
	
	/*if(n<size){
		int tSize = size/n;
		if(tid<(size%n)
			tSize++;
		for(int i; i<tSize; i++)
			if(sdata[tid]<num[glo
	}
	*/
	__syncthreads();
	
	//get a block max by performing a tree-structured 
	//reduction akin to that depicted in slide 18 of 
	//the lecture 8 pp
	
	for (int stride = blockDim.x / 2; stride > 0; stride = stride / 2) {
        if (tid < stride) {
            if (sdata[tid] < sdata[tid + stride]) {
                sdata[tid] = sdata[tid + stride];
            }
        }
        __syncthreads();
	}
	
	if(tid==0){//store the block maxes in global memory
		num[blockIdx.x]=sdata[0];
	}
}

int main(int argc, char *argv[])
{
	hipDeviceProp_t prop;
	hipError_t propErr = hipGetDeviceProperties(&prop, 0);
	
	
	if (propErr != hipSuccess) {
		printf("unable to get device properties\n");
	}
	
	
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
	unsigned int* cudaNumbers;
	unsigned int block;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);
	block = size/thread;
    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

	

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
	   
	//for(int i = 0; i < size; i++) {
	//	printf("%d ", numbers[i]);
    //}
    //printf("\n"); 
	 
	hipMalloc((void**)&cudaNumbers, (size * sizeof(unsigned int)));
	hipMemcpy(cudaNumbers, numbers, (size * sizeof(unsigned int)), hipMemcpyHostToDevice);
	
	unsigned int cudaSize=size;
	
	unsigned int cudaN = block*thread;		
	
	getmaxcu<<<block, thread>>>(cudaNumbers, cudaSize, cudaN);  
	getmaxcu<<<1, block>>>(cudaNumbers, block, block);
	
	hipMemcpy(numbers, cudaNumbers, sizeof(unsigned int), hipMemcpyDeviceToHost);//only copies back the max, which should be in the first element of the array
	printf(" The maximum number in the array is: %u\n", numbers[0]);

    free(numbers);
	hipFree(cudaNumbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/