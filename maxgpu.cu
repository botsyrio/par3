#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

hipDeviceProp_t prop;
hipError_t error = hipGetDeviceProperties(&prop, 0);


unsigned int getmax(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
	printf("char name[256]; = %s\n
              size_t totalGlobalMem; = %d\n
              size_t sharedMemPerBlock; = %d\n
              int regsPerBlock; = %d\n
              int warpSize; = %d\n
              size_t memPitch; = %d\n
              int maxThreadsPerBlock; = %d\n
              int maxThreadsDim[3]; = %d\n
              int maxGridSize[3]; = %d\n
              int clockRate; = %d\n
              size_t totalConstMem; = %d\n
              int major; = %d\n
              int minor; = %d\n
              size_t textureAlignment; = %d\n
              size_t texturePitchAlignment; = %d\n
              int deviceOverlap; = %d\n
              int multiProcessorCount; = %d\n
              int kernelExecTimeoutEnabled; = %d\n
              int integrated; = %d\n
              int canMapHostMemory; = %d\n
              int computeMode; = %d\n
              int maxTexture1D; = %d\n
              int maxTexture1DMipmap; = %d\n
              int maxTexture1DLinear; = %d\n
              int maxTexture2D[2]; = %d\n
              int maxTexture2DMipmap[2]; = %d\n
              int maxTexture2DLinear[3]; = %d\n
              int maxTexture2DGather[2]; = %d\n
              int maxTexture3D[3]; = %d\n
              int maxTexture3DAlt[3]; = %d\n
              int maxTextureCubemap; = %d\n
              int maxTexture1DLayered[2]; = %d\n
              int maxTexture2DLayered[3]; = %d\n
              int maxTextureCubemapLayered[2]; = %d\n
              int maxSurface1D; = %d\n
              int maxSurface2D[2]; = %d\n
              int maxSurface3D[3]; = %d\n
              int maxSurface1DLayered[2]; = %d\n
              int maxSurface2DLayered[3]; = %d\n
              int maxSurfaceCubemap; = %d\n
              int maxSurfaceCubemapLayered[2] = %d\n;
              size_t surfaceAlignment; = %d\n
              int concurrentKernels; = %d\n
              int ECCEnabled; = %d\n
              int pciBusID; = %d\n
              int pciDeviceID; = %d\n
              int pciDomainID; = %d\n
              int tccDriver; = %d\n
              int asyncEngineCount; = %d\n
              int unifiedAddressing; = %d\n
              int memoryClockRate; = %d\n
              int memoryBusWidth; = %d\n
              int l2CacheSize; = %d\n
              int maxThreadsPerMultiProcessor; = %d\n
              int streamPrioritiesSupported; = %d\n
              int globalL1CacheSupported; = %d\n
              int localL1CacheSupported; = %d\n
              size_t sharedMemPerMultiprocessor; = %d\n
              int regsPerMultiprocessor; = %d\n
              int managedMemSupported; = %d\n
              int isMultiGpuBoard; = %d\n
              int multiGpuBoardGroupID; = %d\n
              int singleToDoublePrecisionPerfRatio; = %d\n
              int pageableMemoryAccess; = %d\n
              int concurrentManagedAccess; = %d\n
              int computePreemptionSupported; = %d\n
              int canUseHostPointerForRegisteredMem; = %d\n
              int cooperativeLaunch; = %d\n
              int cooperativeMultiDeviceLaunch;", prop->name,
              prop->totalGlobalMem,
              prop->sharedMemPerBlock,
            prop->regsPerBlock,
              prop->warpSize,
              prop->memPitch,
              prop->maxThreadsPerBlock,
              prop->maxThreadsDim[3],
              prop->maxGridSize[3],
              prop->clockRate,
              prop->totalConstMem,
              prop->major,
              prop->minor,
              prop->textureAlignment,
             prop->texturePitchAlignment,
              prop->deviceOverlap,
              prop->multiProcessorCount,
              prop->kernelExecTimeoutEnabled,
            prop->integrated,
              prop->canMapHostMemory,
              prop-> computeMode,
              prop-> maxTexture1D,
              prop-> maxTexture1DMipmap,
              prop-> maxTexture1DLinear,
              prop-> maxTexture2D[2],
              prop-> maxTexture2DMipmap[2],
              prop-> maxTexture2DLinear[3],
              prop-> maxTexture2DGather[2],
              prop-> maxTexture3D[3],
              prop-> maxTexture3DAlt[3],
              prop-> maxTextureCubemap,
              prop-> maxTexture1DLayered[2],
              prop-> maxTexture2DLayered[3],
              prop-> maxTextureCubemapLayered[2],
              prop-> maxSurface1D,
              prop-> maxSurface2D[2],
              prop-> maxSurface3D[3],
              prop-> maxSurface1DLayered[2],
              prop-> maxSurface2DLayered[3],
              prop-> maxSurfaceCubemap,
              prop-> maxSurfaceCubemapLayered[2],
              prop-> surfaceAlignment,
              prop-> concurrentKernels,
              prop-> ECCEnabled,
              prop-> pciBusID,
              prop-> pciDeviceID,
              prop-> pciDomainID,
              prop-> tccDriver,
              prop-> asyncEngineCount,
              prop->unifiedAddressing,
              prop->memoryClockRate,
              prop->memoryBusWidth,
              prop->l2CacheSize,
              prop->maxThreadsPerMultiProcessor,
              prop->streamPrioritiesSupported,
              prop->globalL1CacheSupported,
              prop-> localL1CacheSupported,
              prop-> sharedMemPerMultiprocessor,
              prop-> regsPerMultiprocessor,
              prop-> managedMemSupported,
              prop-> isMultiGpuBoard,
              prop-> multiGpuBoardGroupID,
              prop-> singleToDoublePrecisionPerfRatio,
              prop-> pageableMemoryAccess,
              prop-> concurrentManagedAccess,
              prop-> computePreemptionSupported,
              prop-> canUseHostPointerForRegisteredMem,
              prop-> cooperativeLaunch,
              prop-> cooperativeMultiDeviceLaunch);
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
	unsigned int block;
	unsigned int thread;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
	   
   getCuda
   // printf(" The maximum number in the array is: %u\n", 
     //      getmax<<<block, thread>>>(cudaNumbers, cudaSize, cudaN));

    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
_global_ unsigned int getmax(unsigned int num[], unsigned int size, int n)
{
	if(id<size%n)
		starting = (threadId.x+blockId.x*blockDim.x)(size/n+1);
	else
		starting = (size%n)(size/n+1)+((threadId.x+blockId.x*blockDim.x)-size%n)(size/n);
	blockId.x*blockDim.x+threadId.x;
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}