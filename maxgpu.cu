#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define thread 512
/*
This code was developed and tested on cuda1

Device 0:
name: GeForce GTX TITAN Black
Compute capability 3.5
total global memory(KB): 6228288
shared mem per block: 49152
regs per block: 65536
warp size: 32
max threads per block: 1024
max thread dim z:1024 y:1024 x:64
max grid size z:2147483647 y:65535 x:65535
clock rate(KHz):
total constant memory (bytes): 65536
multiprocessor count 15
integrated: 0
async engine count: 1
memory bus width: 384
memory clock rate (KHz): 3500000
L2 cache size (bytes): 1572864
max threads per SM: 2048

*/


__global__ void getmaxcu(unsigned int num[], unsigned int size, int n){
	/*if(id<size%n)
		starting = (threadIdx.x+blockId.x*blockDim.x)(size/n+1);
	else
		starting = (size%n)(size/n+1)+((threadId.x+blockId.x*blockDim.x)-size%n)(size/n);*/
		
	
	unsigned int tid = threadIdx.x;
	unsigned int gloid = blockIdx.x*blockDim.x+threadIdx.x;	
	//unsigned int tSize = size/n;
	
	//const unsigned int dim = blockDim.x;
	__shared__ int sdata[thread]; // shared data
	
	//if(tid<size%n)
	//	tSize++;
	//__syncthreads();	
	
	//each thread iterates over its section of the large array
	sdata[tid]=num[gloid];
	//for(unsigned int i = 0; i < tSize; i++)
	//	if(sdata[tid]<num[gloid+i])
	//		sdata[tid]=num[gloid+i];
	if(gloid>=size){
		sdata[threadIdx.x]=0;
	}
			
	__syncthreads();
	
	//get a block max by performing a tree-structured 
	//reduction akin to that depicted in slide 17 of 
	//the lecture 8 pp
	
	for (int stride = blockDim.x / 2; stride > 0; stride = stride / 2) {
        if (threadIdx.x < stride) {
            if (sdata[threadIdx.x] < sdata[threadIdx.x + stride]) {
                sdata[threadIdx.x] = sdata[threadIdx.x + stride];
            }
        }
        __syncthreads();
	}
	
	/*for(unsigned int stride = 1; stride<blockDim.x; stride*=2){
		if(tid%(2*stride)==0){
			if(sdata[tid]<sdata[tid+stride])
				sdata[tid]=sdata[tid+stride];
		}
		__syncthreads();
	}*/
	
	
	/*
	//BELOW: version that does not use shared memory
	
	if(tid<size%n)
		tSize++;
	__syncthreads();	
	
	//each thread iterates over its section of the large array
	for(unsigned int i = 0; i < tSize; i++)
		if(num[gloid]<num[gloid+i])
			num[gloid]=num[gloid+i];
			
	__syncthreads();
	
	//get a block max by performing a tree-structured 
	//reduction akin to that depicted in slide 17 of 
	//the lecture 8 pp
	
	for(unsigned int stride = 1; stride<blockDim.x; stride*=2){
		if(tid%(2*stride)==0){
			if(sdata[tid]<sdata[tid+stride])
				sdata[tid]=sdata[tid+stride];
		}
		__syncthreads();
	}
	*/
	
	if(tid==0){//store the block maxes in global memory
		num[blockIdx.x]=sdata[0];
	}
	

	//return(num[0]);

}

int main(int argc, char *argv[])
{
	hipDeviceProp_t prop;
	hipError_t propErr = hipGetDeviceProperties(&prop, 0);
	
	
	if (propErr != hipSuccess) {
		printf("unable to get device properties\n");
	}
	
	
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
	unsigned int* cudaNumbers;
	unsigned int block;
	
	block = 30;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

	

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
	   
	for(int i = 0; i < size; i++) {
		printf("%d ", numbers[i]);
    }
    printf("\n"); 
	 
	hipMalloc((void**)&cudaNumbers, (size * sizeof(unsigned int)));
	hipMemcpy(cudaNumbers, numbers, (size * sizeof(unsigned int)), hipMemcpyHostToDevice);
	
	unsigned int cudaSize=size;
	//cudaMalloc((void**)&cudaSize, sizeof(unsigned int));
	//cudaMemcpy(cudaSize, size, (sizeof(unsigned int)), cudaMemcpyHostToDevice);
	
	
	unsigned int cudaN = block*thread;
	//unsigned int cudaN;
	//cudaMalloc((void**)&cudaN, sizeof(unsigned int));
	//cudaMemcpy(cudaN, n, (sizeof(unsigned int)), cudaMemcpyHostToDevice);
		
	
	getmaxcu<<<block, thread, sizeof(unsigned int)*thread>>>(cudaNumbers, cudaSize, cudaN);
	//cudaSize/thread;
   
    
	getmaxcu<<<1, block, sizeof(unsigned int)*block>>>(cudaNumbers, block, block);
	printf("%s\n", hipMemcpy(numbers, cudaNumbers, size*sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	for(int i = 0; i < size; i++) {
		printf("%d ", numbers[i]);
    }
    printf("\n"); 
	printf(" The maximum number in the array is: %u\n", numbers[0]);

    free(numbers);
	hipFree(cudaNumbers);
	//cudaFree(cudaSize);
	//cudaFree(cudaN);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/